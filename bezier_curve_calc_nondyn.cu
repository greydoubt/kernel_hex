#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_TESS_POINTS 32 05
//A structure containing all parameters needed to tessellate a Bezier line

struct BezierLine {
  float2 CP[3]; //Control points for the line
  float2 vertexPos[MAX_TESS_POINTS]; //Vertex position array to tessellate into
  int nVertices; //Number of tessellated vertices
};

__global__ void computeBezierLines(BezierLine *bLines, int nLines) {

  int bidx = blockIdx.x;
  if(bidx < nLines){

  //Compute the curvature of the line
  float curvature = computeCurvature(bLines);
  //From the curvature, compute the number of tessellation points
  int nTessPoints = min(max((int)(curvature*16.0f),4),32); bLines[bidx].nVertices = nTessPoints;
  //Loop through vertices to be tessellated, incrementing by blockDim.x
  for(int inc = 0; inc < nTessPoints; inc += blockDim.x){
    int idx = inc + threadIdx.x; //Compute a unique index for this point if(idx < nTessPoints){
    float u = (float)idx/(float)(nTessPoints-1); //Compute u from idx float omu = 1.0f - u; //pre-compute one minus u
    float B3u[3]; //Compute quadratic Bezier coefficients
    B3u[0] = omu*omu;
    B3u[1] = 2.0f*u*omu;
    B3u[2] = u*u;
    float2 position = {0,0}; //Set position to zero for(int i = 0; i < 3; i++){
//Add the contribution of the i'th control point to position
    position = position + B3u[i] * bLines[bidx].CP[i]; }
//Assign value of vertex position to the correct array element
    bLines[bidx].vertexPos[idx] = position; }
}

#define N_LINES 256
#define BLOCK_DIM 32

int main( int argc, char **argv ) {
//Allocate and initialize array of lines in host memory
BezierLine *bLines_h = new BezierLine[N_LINES];
initializeBLines(bLines_h);

//Allocate device memory for array of Bezier lines
BezierLine *bLines_d;
hipMalloc((void**)&bLines_d, N_LINES*sizeof(BezierLine));
hipMemcpy(bLines_d,bLines_h, N_LINES*sizeof(BezierLine),hipMemcpyHostToDevice);

  //Call the kernel to tessellate the lines
computeBezierLines<<<N_LINES, BLOCK_DIM>>>(bLines_d, N_LINES );

hipFree(bLines_d); //Free the array of lines in device memory
delete[] bLines_h; //Free the array of lines in host memory
}
