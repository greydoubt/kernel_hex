#include "hip/hip_runtime.h"
__global__ void cmpMu(float* rPhi, iPhi, rD, iD, rMu, iMu) {
int m = blockIdx.x*MU_THREAEDS_PER_BLOCK + threadIdx.x;
        rMu[m] = rPhi[m]*rD[m] + iPhi[m]*iD[m];
        iMu[m] = rPhi[m]*iD[m] – iPhi[m]*rD[m];
      }
