#include "hip/hip_runtime.h"
__global__ void histogram_privatized_kernel(
  unsigned char* input, 
  unsigned int* bins, 
  unsigned int num_elements, 
  unsigned int num_bins
) { 
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
// Privatized bins
  extern __shared__ unsigned int histo_s[];

  for(unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx +=blockDim.x) {
  histo_s[binIdx] = 0u; 
    }
__syncthreads();
  
  // Histogram
for (unsigned int i = tid; i < num_elements; i += blockDim.x*gridDim.x) {
  int alphabet_position = buffer[i] – “a”;
  if (alphabet_position >= 0 && alpha_position < 26) atomicAdd(&(histo_s[alphabet_position/4]), 1);
} 
__syncthreads();
  
// Commit to global memory
for(unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
  atomicAdd(&(histo[binIdx]), histo_s[binIdx]);
} }
